
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define BLOCK_SIZE 256

__global__ void type1(int n, double lr, double lambda, double * W) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i=index;i<n;i+=stride) {
        W[i] = (1.0 - lr* lambda) * W[i];
    }
}

__global__ void type2(int n, double lr, double lambda, double * W, int rand_choice, double * X, double * Y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i=index;i<n;i+=stride) {
        W[i] = (1.0 - lr* lambda) * W[i] + (lr * Y[rand_choice])*X[rand_choice * n + i];
    }
}

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
	    old = atomicCAS(address_as_ull, assumed,
        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void dot(int n, double * W, double *X, int rand_choice, double * res) {
   __shared__ double temp[BLOCK_SIZE];
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index < n)   temp[threadIdx.x] = W[index] * X[rand_choice * n + index];
   else             temp[threadIdx.x] = 0;
   __syncthreads();
   if (threadIdx.x == 0){
       double sum = 0;
       for (int i=0;i<BLOCK_SIZE;i++) sum += temp[i];
       atomicAddDouble(res, sum);
   }
}


int main() {
    srand(time(NULL));
    ifstream  trainfile ("train.txt");
    ifstream labelfile ("labels.txt");
    int n_samples=200;
    int n_features=50000;

    double *W, *X, *Y, *res;
    double *d_W, *d_X, *d_Y, *d_res;
    hipEvent_t start, stop;
    float elapsedTime;
    
    W = (double *) malloc(n_features * sizeof(double));
    X = (double *) malloc(n_samples * n_features * sizeof(double));
    Y = (double *) malloc(n_samples * sizeof(double));
    res = (double *) malloc(sizeof(double));

    hipMalloc(&d_W, n_features * sizeof(double));
    hipMalloc(&d_X, n_samples * n_features * sizeof(double));
    hipMalloc(&d_Y, n_samples * sizeof(double));
    hipMalloc(&d_res, sizeof(double));
    
    for (int i=0;i<n_samples;i++) {
		for (int j=0;j<n_features;j++)
			trainfile >> X[i*n_features + j];
    }
	for (int i=0;i<n_samples;i++) {
		labelfile >> Y[i];
		if (Y[i] == 0) {
			Y[i] = -1;
		}
	}
    for (int i=0;i<n_features;i++)  W[i] = 0;

    hipMemcpy(d_X, X, n_samples * n_features * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, n_features * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, n_samples * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, n_features * sizeof(double), hipMemcpyHostToDevice);

    int num_iters = 100;
    double lambda = 1.0;
    hipEventCreate(&start);
    hipEventRecord(start,0);
    for (int iters=1;iters<=num_iters;iters++) {
    	int numBlocks = (n_features + BLOCK_SIZE - 1) / BLOCK_SIZE;
        double lr = 1.0 / (lambda * iters);
        int rand_choice = rand() % n_samples;
        cout << rand_choice << endl;
 	   	*res = 0;
 	    hipMemcpy(d_res, res, sizeof(double), hipMemcpyHostToDevice);
 	    dot<<<numBlocks, BLOCK_SIZE>>>(n_features, d_W, d_X, rand_choice, d_res);
 	    hipMemcpy(res, d_res, sizeof(double), hipMemcpyDeviceToHost);
	    if (Y[rand_choice] * res[0] >= 1.0)
	        type1<<<numBlocks, BLOCK_SIZE>>>(n_features, lr, lambda, d_W);
	    else
	        type2<<<numBlocks, BLOCK_SIZE>>>(n_features, lr, lambda, d_W, rand_choice, d_X, d_Y);
        hipMemcpy(W, d_W, n_features * sizeof(double), hipMemcpyDeviceToHost);
    }
    hipMemcpy(W, d_W, n_features * sizeof(double), hipMemcpyDeviceToHost);
    
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start,stop);

    cout << "Train time " << elapsedTime << endl;
    double correct = 0.0;
    for (int i=0;i<n_samples;i++) {
        double val = 0.0;
        for (int j=0;j<n_features;j++)
            val += W[j] * X[i * n_features + j];
        if (val * Y[i] >= 0)
            correct += 1;
    }
    cout << "Correct " << correct << endl;
    printf("Accuracy %lf\n", correct / n_samples);
    return 0;
}
