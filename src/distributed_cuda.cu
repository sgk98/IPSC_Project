
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define BLOCK_SIZE 256

__global__ void type1(int n, double lr, double lambda, double * W) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i=index;i<n;i+=stride) {
        W[i] = (1.0 - lr* lambda) * W[i];
    }
}

__global__ void type2(int n, double lr, double lambda, double * W, int rand_choice, double * X, double * Y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i=index;i<n;i+=stride) {
        W[i] = (1.0 - lr* lambda) * W[i] + (lr * Y[rand_choice])*X[rand_choice * n + i];
    }
}

__device__ double atomicAddDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
	    old = atomicCAS(address_as_ull, assumed,
        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void dot(int n, double * W, double *X, int rand_choice, double * res) {
   __shared__ double temp[BLOCK_SIZE];
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index < n)   temp[threadIdx.x] = W[index] * X[rand_choice * n + index];
   else             temp[threadIdx.x] = 0;
   __syncthreads();
   if (threadIdx.x == 0){
       double sum = 0;
       for (int i=0;i<BLOCK_SIZE;i++) sum += temp[i];
       atomicAddDouble(res, sum);
   }
}

__global__ void pegasos_per_thread(int num_samples, int num_features, double * W, double * X, double * Y, double lambda, int num_iters, double * random_arr,  int k, double * test) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int n_samples_per_thread = num_samples / k;
    for(int iters=1; iters<=num_iters; iters++)
    {
        double lr = 1.0 / (lambda * iters);
        int rand_choice = random_arr[iters];
        double pred_output = 0;
        for (int i=0; i<num_features; i++)
            pred_output += W[index * num_features + i] * X[(n_samples_per_thread * index + rand_choice) * num_features + i];
        if (Y[rand_choice] * pred_output >= 1.0) {
            for (int i=0; i<num_features; i++)
                W[index * num_features + i] = (1.0 - lr * lambda) * W[index * num_features + i];
        } else {
            for (int i=0; i<num_features; i++)
                W[index * num_features + i] = (1.0 - lr * lambda) * W[index * num_features + i] + (lr * Y[rand_choice]) * X[(n_samples_per_thread * index + rand_choice) * num_features + i];
        }
    }
}


int main() {
    srand(time(NULL));
    ifstream  trainfile ("train.txt");
    ifstream labelfile ("labels.txt");
    int n_samples=20000;
    int n_features=500;
    int k = 1000;
    int numBlocks = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int num_iters = 100;
    double lambda = 1.0;
    int n_samples_per_thread = n_samples / k;

    double *W, *X, *Y, *final_W, *random_arr, *test;
    double *d_W, *d_X, *d_Y, *d_random_arr, *d_test;

    W = (double *) malloc(k * n_features * sizeof(double));
    final_W = (double *) malloc(n_features * sizeof(double));
    X = (double *) malloc(n_samples * n_features * sizeof(double));
    Y = (double *) malloc(n_samples * sizeof(double));
    random_arr = (double *) malloc(num_iters * sizeof(double));
    test = (double *) malloc(sizeof(double));

    hipMalloc(&d_W, k * n_features * sizeof(double));
    hipMalloc(&d_X, n_samples * n_features * sizeof(double));
    hipMalloc(&d_Y, n_samples * sizeof(double));
    hipMalloc(&d_random_arr, num_iters * sizeof(double));
    hipMalloc(&d_test, sizeof(double));
    
    for (int i=0;i<n_samples;i++) {
		for (int j=0;j<n_features;j++)
			trainfile >> X[i*n_features + j];
    }
	for (int i=0;i<n_samples;i++) {
		labelfile >> Y[i];
		if (Y[i] == 0) {
			Y[i] = -1;
		}
	}
    for (int i=0;i<k;i++)  {
        for (int j=0;j<n_features;j++)   W[i * n_features + j] = 0;
    }
    for (int i=0;i<n_features;i++)  final_W[i] = 0;
    for (int i=0;i<num_iters;i++)   random_arr[i] = rand() % n_samples_per_thread;   

    hipMemcpy(d_X, X, n_samples * n_features * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, k * n_features * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, n_samples * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_random_arr, random_arr, num_iters * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_test, test, sizeof(double), hipMemcpyHostToDevice);
    pegasos_per_thread<<<numBlocks, BLOCK_SIZE>>>(n_samples, n_features, d_W, d_X, d_Y, lambda, num_iters, d_random_arr, k, d_test);
    hipMemcpy(W, d_W, k * n_features * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(test, d_test, sizeof(double), hipMemcpyDeviceToHost);
    
    for (int i=0;i<k;i++)
        for (int j=0;j<n_features;j++)
            cout << W[i * n_features + j]  << " ";
    for (int i=0;i<k;i++) {
        for (int j=0;j<n_features;j++)  final_W[j] += (W[i * n_features + j]);
    }
    cout << "\nFinalW\n";
    for (int i=0;i<n_features;i++)  {final_W[i] /= k;   cout << final_W[i] << " ";}
    cout << "test " << test[0] << endl;

    double correct = 0.0;
    for (int i=0;i<n_samples;i++) {
        double val = 0.0;
        for (int j=0;j<n_features;j++)
            val += final_W[j] * X[i * n_features + j];
        if (val * Y[i] >= 0)
            correct += 1;
    }
    cout << "Correct " << correct << endl;
    printf("Accuracy %lf\n", correct / n_samples);
    return 0;
}
