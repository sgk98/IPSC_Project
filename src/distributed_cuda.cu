
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define BLOCK_SIZE 256

__global__ void pegasos_per_thread(int num_samples, int num_features, double * W, double * X, double * Y, double lambda, int num_iters, double * random_arr,  int k) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int n_samples_per_thread = num_samples / k;
    for(int iters=1; iters<=num_iters; iters++)
    {
        double lr = 1.0 / (lambda * iters);
        int rand_choice = random_arr[iters];
        double pred_output = 0;
        for (int i=0; i<num_features; i++)
            pred_output += W[index * num_features + i] * X[(n_samples_per_thread * index + rand_choice) * num_features + i];
        if (Y[rand_choice] * pred_output >= 1.0) {
            for (int i=0; i<num_features; i++)
                W[index * num_features + i] = (1.0 - lr * lambda) * W[index * num_features + i];
        } else {
            for (int i=0; i<num_features; i++)
                W[index * num_features + i] = (1.0 - lr * lambda) * W[index * num_features + i] + (lr * Y[rand_choice]) * X[(n_samples_per_thread * index + rand_choice) * num_features + i];
        }
    }
}


int main() {
    srand(time(NULL));
    ifstream  trainfile ("train.txt");
    ifstream labelfile ("labels.txt");
    int n_samples=20000;
    int n_features=500;
    int k = 1000;
    int numBlocks = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int num_iters = 100;
    double lambda = 1.0;
    int n_samples_per_thread = n_samples / k;

    double *W, *X, *Y, *final_W, *random_arr;
    double *d_W, *d_X, *d_Y, *d_random_arr;

    W = (double *) malloc(k * n_features * sizeof(double));
    final_W = (double *) malloc(n_features * sizeof(double));
    X = (double *) malloc(n_samples * n_features * sizeof(double));
    Y = (double *) malloc(n_samples * sizeof(double));
    random_arr = (double *) malloc(num_iters * sizeof(double));

    hipMalloc(&d_W, k * n_features * sizeof(double));
    hipMalloc(&d_X, n_samples * n_features * sizeof(double));
    hipMalloc(&d_Y, n_samples * sizeof(double));
    hipMalloc(&d_random_arr, num_iters * sizeof(double));
    
    for (int i=0;i<n_samples;i++) {
		for (int j=0;j<n_features;j++)
			trainfile >> X[i*n_features + j];
    }
	for (int i=0;i<n_samples;i++) {
		labelfile >> Y[i];
		if (Y[i] == 0) {
			Y[i] = -1;
		}
	}
    for (int i=0;i<k;i++)  {
        for (int j=0;j<n_features;j++)   W[i * n_features + j] = 0;
    }
    for (int i=0;i<n_features;i++)  final_W[i] = 0;
    for (int i=0;i<num_iters;i++)   random_arr[i] = rand() % n_samples_per_thread;   

    hipMemcpy(d_X, X, n_samples * n_features * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, k * n_features * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, n_samples * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_random_arr, random_arr, num_iters * sizeof(double), hipMemcpyHostToDevice);
    pegasos_per_thread<<<numBlocks, BLOCK_SIZE>>>(n_samples, n_features, d_W, d_X, d_Y, lambda, num_iters, d_random_arr, k);
    hipMemcpy(W, d_W, k * n_features * sizeof(double), hipMemcpyDeviceToHost);
    
    // for (int i=0;i<k;i++)
        // for (int j=0;j<n_features;j++)
            // cout << W[i * n_features + j]  << " ";
    for (int i=0;i<k;i++) {
        for (int j=0;j<n_features;j++)  final_W[j] += (W[i * n_features + j]);
    }
    // cout << "\nFinalW\n";
    // for (int i=0;i<n_features;i++)  {final_W[i] /= k;   cout << final_W[i] << " ";}

    double correct = 0.0;
    for (int i=0;i<n_samples;i++) {
        double val = 0.0;
        for (int j=0;j<n_features;j++)
            val += final_W[j] * X[i * n_features + j];
        if (val * Y[i] >= 0)
            correct += 1;
    }
    cout << "Correct " << correct << endl;
    printf("Accuracy %lf\n", correct / n_samples);
    return 0;
}
